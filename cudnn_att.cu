#include "hip/hip_runtime.h"
// all cudnn-related functions are in this file, so that they don't need to be recompiled everytime
// we change some unrelated piece of the code.
// TODO this currently duplicates some of the utilities from the main file

#include <cudnn_frontend.h>
#include <hip/hip_bf16.h>
#include <nvtx3/nvToolsExt.h>

// Specific configurations based on the enabled precision
#if defined(ENABLE_FP32)
typedef float floatX;

// use fp16 (note: this may require gradient scaler, currently not implemented!)
#elif defined(ENABLE_FP16)
typedef half floatX;
#define CUBLAS_LOWP HIP_R_16F

#else // Default to bfloat16
typedef __hip_bfloat16 floatX;
#endif

// CUDA error checking
static void cudaCheck(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
               hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// Profiler utils
namespace {
    class NvtxRange {
    public:
        NvtxRange(const char* s) { nvtxRangePush(s); }

        NvtxRange(const std::string& base_str, int number) {
            std::string range_string = base_str + " " + std::to_string(number);
            nvtxRangePush(range_string.c_str());
        }

        ~NvtxRange() { nvtxRangePop(); }
    };
}
#define NVTX_RANGE_FN() NvtxRange nvtx_range(__FUNCTION__)

namespace fe = cudnn_frontend;
#if CUBLAS_LOWP == HIP_R_16BF
#define CUDNN_16BIT fe::DataType_t::BFLOAT16
#else
#define CUDNN_16BIT fe::DataType_t::HALF
#endif

static hipdnnHandle_t cudnn_handle;
static size_t cudnn_workspace_size = 0; // dynamically allocated as needed (up to 256MiB!)
static void* cudnn_workspace = NULL;
#define checkCudnnErr(err) assert((int)err == 0);

static void checkCudnnFE(fe::error_object e, const char *file, int line) {
    if(!e.is_good()) {
        printf("[CUDNN ERROR] at file %s:%d:\n%s\n", file, line, e.err_msg.c_str());
        exit(EXIT_FAILURE);
    }
}
#define checkCudnnFE(err) checkCudnnFE(err, __FILE__, __LINE__)

using graph_tensors_fwd = std::tuple<std::shared_ptr<fe::graph::Graph>,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // Q,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // K,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // V,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // Attn_scale,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // O
    std::shared_ptr<fe::graph::Tensor_attributes> // Stats
>;

using graph_tensors_bwd = std::tuple<std::shared_ptr<fe::graph::Graph>,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // Q,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // K,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // V,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // O
    std::shared_ptr<fe::graph::Tensor_attributes>,  // dO
    std::shared_ptr<fe::graph::Tensor_attributes>,  // Stats
    std::shared_ptr<fe::graph::Tensor_attributes>,  // Attn_scale,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // dQ,
    std::shared_ptr<fe::graph::Tensor_attributes>,  // dK,
    std::shared_ptr<fe::graph::Tensor_attributes> // dV
>;

// Need a cache because graph->build_operation_graph() is slow but everything else seems fast
using cache_type_fwd = std::unordered_map<std::size_t, graph_tensors_fwd>;
using cache_type_bwd = std::unordered_map<std::size_t, graph_tensors_bwd>;

// Loosely based on cuDNN frontend samples functions and massively simplified
template <typename... Args>
auto lookup_cache_or_build_graph_fwd(Args... args) {
    static cache_type_fwd user_maintained_cache_fwd;
    auto [B, H, T, HS, is_inference_only] = std::make_tuple(args...);

    auto graph = std::make_shared<fe::graph::Graph>();
    graph->set_io_data_type(CUDNN_16BIT)
        .set_intermediate_data_type(fe::DataType_t::FLOAT)
        .set_compute_data_type(fe::DataType_t::FLOAT);

    // QKV is (B, T, 3, NH, HS) which cuDNN can handle directly without an external permute
    auto Q = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("Q")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T,  HS, 3 * H * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("K")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("V")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes()
                                        .set_name("attn_scale")
                                        .set_dim({1, 1, 1, 1})
                                        .set_stride({1, 1, 1, 1})
                                        .set_is_pass_by_value(true)
                                        .set_data_type(fe::DataType_t::FLOAT));

    auto sdpa_options = fe::graph::SDPA_attributes().set_name("flash_attention");
    sdpa_options.set_is_inference(is_inference_only);
    sdpa_options.set_attn_scale(attn_scale);
    sdpa_options.set_causal_mask(true);

    // Create the graph operation and get the output tensors back
    auto [O, stats] = graph->sdpa(Q, K, V, sdpa_options);

    // Output is (B, T, NH, HS) BF16/FP16 and stats for backward pass is (B, NH, T) FP32
    O->set_output(true).set_dim({B, H, T, HS}).set_stride({H * HS * T, HS, H * HS, 1});

    assert(stats == nullptr || is_inference_only == false);
    if (is_inference_only == false) {
        stats->set_output(true).set_data_type(fe::DataType_t::FLOAT)
            .set_dim({B, H, T, 1})
            .set_stride({H * T, T, 1, 1});
    }

    checkCudnnFE(graph->validate());
    auto key = graph->key();
    auto it = user_maintained_cache_fwd.find(key);
    if (it != user_maintained_cache_fwd.end()) {
        return it->second;
    }

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    checkCudnnFE(graph->build_operation_graph(cudnn_handle));
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    checkCudnnFE(graph->check_support(cudnn_handle));
    checkCudnnFE(graph->build_plans(cudnn_handle));

    auto tuple = std::make_tuple(graph, Q, K, V, attn_scale, O, stats);
    user_maintained_cache_fwd.insert({key, tuple});
    return tuple;
}

template <typename... Args>
auto lookup_cache_or_build_graph_bwd(Args... args) {
    static cache_type_bwd user_maintained_cache_bwd;
    auto [B, NH, T, HS] = std::make_tuple(args...);

    auto graph = std::make_shared<fe::graph::Graph>();
    graph->set_io_data_type(CUDNN_16BIT)
        .set_intermediate_data_type(fe::DataType_t::FLOAT)
        .set_compute_data_type(fe::DataType_t::FLOAT);

    // (B, N, 3, NH, HS)
    // must come from inp (which means we also need to convert THAT to FP16)
    auto Q = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("Q")
                               .set_dim({B, NH, T, HS})
                               .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("K")
                               .set_dim({B, NH, T, HS})
                               .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("V")
                               .set_dim({B, NH, T, HS})
                               .set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1}));
    auto O = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("O")
                               .set_dim({B, NH, T, HS})
                               .set_stride({NH * HS * T, HS, NH * HS, 1}));
    auto dO = graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("dO")
                                .set_dim({B, NH, T, HS})
                                .set_stride({NH * HS * T, HS, NH * HS, 1}));

    auto stats = graph->tensor(fe::graph::Tensor_attributes()
                                   .set_name("stats")
                                   .set_dim({B, NH, T, 1})
                                   .set_stride({NH * T, T, 1, 1})
                                   .set_data_type(fe::DataType_t::FLOAT));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes()
                                        .set_name("attn_scale")
                                        .set_dim({1, 1, 1, 1})
                                        .set_stride({1, 1, 1, 1})
                                        .set_is_pass_by_value(true)
                                        .set_data_type(fe::DataType_t::FLOAT));
    auto sdpa_backward_options = fe::graph::SDPA_backward_attributes()
        .set_name("flash_attention_backward")
        .set_causal_mask(true)
        .set_attn_scale(attn_scale);

    // Create the graph operation and get the output tensors back
    auto [dQ, dK, dV] = graph->sdpa_backward(Q, K, V, O, dO, stats, sdpa_backward_options);

    dQ->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1});
    dK->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1});
    dV->set_output(true).set_dim({B, NH, T, HS}).set_stride({3 * NH * HS * T, HS, 3 * NH * HS, 1});

    checkCudnnFE(graph->validate());
    auto key = graph->key();
    auto it = user_maintained_cache_bwd.find(key);
    if (it != user_maintained_cache_bwd.end()) {
        return it->second;
    }

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    checkCudnnFE(graph->build_operation_graph(cudnn_handle));
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    checkCudnnFE(graph->check_support(cudnn_handle));
    checkCudnnFE(graph->build_plans(cudnn_handle));

    auto tuple = std::make_tuple(graph, Q, K, V, O, dO, stats, attn_scale, dQ, dK, dV);
    user_maintained_cache_bwd.insert({key, tuple});
    return tuple;
}

void attention_forward_cudnn(floatX* out,  // output: (B, T, NH, HS)
                             float* stats, // output for backward pass: (B, NH, T)
                             floatX* inp,  // input: (B, T, 3, NH, HS) QKV
                             int B, int T, int NH, int C) {
    NVTX_RANGE_FN();
    int HS = C / NH; // number of features per head
    bool is_inference_only = (stats == nullptr);

    // Get graph and tensors from cache (or generate it on first use)
    auto [graph, Q, K, V, attn_scale, O, softmax_stats] =
        lookup_cache_or_build_graph_fwd(B, NH, T, HS, is_inference_only);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = inp;
    void* devPtrK = (inp + C);
    void* devPtrV = (inp + 2 * C);
    float attn_scale_cpu = 1.0 / sqrtf(HS);
    void* devPtrO = out;

    // Build variant pack
    std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void*> variant_pack = {
        {Q, devPtrQ}, {K, devPtrK}, {V, devPtrV}, {attn_scale, &attn_scale_cpu}, {O, devPtrO}};

    // Add the stats tensor unless we are only doing inference (only needed for backward pass)
    if (is_inference_only == false) {
        variant_pack[softmax_stats] = stats;
    }

    // Reallocate the workspace if the required size is greater than the current workspace
    // By default, cuDNN uses up to 256MiB of workspace, so we don't want to just allocate the maximum
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    // Execute graph
    checkCudnnFE(graph->execute(cudnn_handle, variant_pack, cudnn_workspace));
    cudaCheck(hipGetLastError());
}

void attention_backward_cudnn(floatX* dqkvr,                                       // output
                              floatX* dout, floatX* qkvr, floatX* o, float* stats, // inputs
                              int B, int T, int NH, int C) {
    NVTX_RANGE_FN();
    int HS = C / NH; // number of features per head

    // Get graph and tensors from cache (or generate it on first use)
    auto [graph, Q, K, V, O, dO, Stats, attn_scale, dQ, dK, dV] =
        lookup_cache_or_build_graph_bwd(B, NH, T, HS);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = qkvr;
    void* devPtrK = (qkvr + NH * HS);
    void* devPtrV = (qkvr + 2 * NH * HS);
    void* devPtrO = o;
    void* devPtrdO = dout;
    void* devPtrStats = stats;
    float attn_scale_cpu = 1.0 / sqrtf(HS);

    void* devPtrdQ = dqkvr;
    void* devPtrdK = (dqkvr + NH * HS);
    void* devPtrdV = (dqkvr + 2 * NH * HS);

    // Build variant pack that links each tensor to its data pointer
    std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void*> variant_pack = {
        {Q, devPtrQ}, {K, devPtrK}, {V, devPtrV}, {O, devPtrO}, {dO, devPtrdO}, {Stats, devPtrStats},
        {dQ, devPtrdQ}, {dK, devPtrdK}, {dV, devPtrdV},
        {attn_scale, &attn_scale_cpu}};

    // Reallocate the workspace if the required size is greater than the current workspace
    // By default, cuDNN uses up to 256MiB of workspace, so we don't want to just allocate the maximum
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    // Execute graph
    checkCudnnFE(graph->execute(cudnn_handle, variant_pack, cudnn_workspace));
    cudaCheck(hipGetLastError());
}

void create_cudnn() {
    checkCudnnErr(hipdnnCreate(&cudnn_handle));
}

void destroy_cudnn() {
    if (cudnn_workspace != NULL) { cudaCheck(hipFree(cudnn_workspace)); }
    checkCudnnErr(hipdnnDestroy(cudnn_handle));
}